
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#define BLOCK_SIZE 8

#define STR_SIZE 256

#define DEVICE 1


#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

void run(int argc, char** argv);


int J, K, L;
int* data;
int** space2D;
int*** space3D;
#define M_SEED 9
int pyramid_height;
int timesteps;

int bornMin = 5, bornMax = 8;
int dieMax = 3, dieMin = 10;

void
init(int argc, char** argv)
{
	if(argc==6){
		J = atoi(argv[1]);
		K = atoi(argv[2]);
                L = atoi(argv[3]);
                timesteps = atoi(argv[4]);
                pyramid_height=atoi(argv[5]);
	}else{
                printf("Usage: cell dim3 dim2 dim1 timesteps pyramid_height\n");
                exit(0);
        }
	data = new int[J*K*L];
        space2D = new int*[J*K];
	space3D = new int**[J];
	for(int n=0; n<J*K; n++)
          space2D[n]=data+L*n;
	for(int n=0; n<J; n++)
          space3D[n]=space2D+K*n;

	int seed = M_SEED;
	srand(seed);

	for (int i = 0; i < J*K*L; i++)
            data[i] = rand()%2;

}

void
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);
}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))
#define EXPAND_RATE 2 // add one iteration will extend the pyramid base by 2 per each borderline
__global__ void evolve(
                int iteration,
                int *gpuDataSrc,
                int *gpuDataDst,
                int J, int K, int L,
                int bornMin, int bornMax,
                int dieMin, int dieMax,
                int blockL, int border)
{
        __shared__ int data[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];

	int bj = blockIdx.x;
	int bk = blockIdx.y/blockL;
	int bl = blockIdx.y%blockL;
	int tj=threadIdx.x;
	int tk=threadIdx.y;
	int tl=threadIdx.z;

        // each block finally computes result for a small block
        // after N iterations.
        // it is the non-overlapping small blocks that cover
        // all the input data

        // calculate the small block size
	int small_block = BLOCK_SIZE-iteration*EXPAND_RATE;

        // calculate the boundary for the block according to
        // the boundary of its small block
        int blkJ = small_block*bj-border;
        int blkK = small_block*bk-border;
        int blkL = small_block*bl-border;

        int blkJmax = blkJ+BLOCK_SIZE-1;
        int blkKmax = blkK+BLOCK_SIZE-1;
        int blkLmax = blkL+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int idxJ = blkJ+tj;
	int idxK = blkK+tk;
	int idxL = blkL+tl;

        int index = idxL+L*(idxK+idxJ*K);

	if(IN_RANGE(idxJ, 0, J-1) &&
          IN_RANGE(idxK, 0, K-1) &&
          IN_RANGE(idxL, 0, L-1) ){
            data[tj][tk][tl] = gpuDataSrc[index];
	}

        // effective range within this block that falls within
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validJmin = (blkJ < 0) ? -blkJ : 0;
        int validJmax = (blkJmax > J-1) ? BLOCK_SIZE-1-(blkJmax-J+1) : BLOCK_SIZE-1;
        int validKmin = (blkK < 0) ? -blkK : 0;
        int validKmax = (blkKmax > K-1) ? BLOCK_SIZE-1-(blkKmax-K+1) : BLOCK_SIZE-1;
        int validLmin = (blkL < 0) ? -blkL : 0;
        int validLmax = (blkLmax > L-1) ? BLOCK_SIZE-1-(blkLmax-L+1) : BLOCK_SIZE-1;

        int isValid =(IN_RANGE(tj, validJmin, validJmax) &&
                      IN_RANGE(tk, validKmin, validKmax) &&
                      IN_RANGE(tl, validLmin, validLmax));

        __syncthreads();

        bool computed=false;
        for (int i=0; i<iteration ; i++){
            computed = false;
            int inLayer =   IN_RANGE(tj, i+1, BLOCK_SIZE-i-2) &&
                            IN_RANGE(tk, i+1, BLOCK_SIZE-i-2) &&
                            IN_RANGE(tl, i+1, BLOCK_SIZE-i-2);

            if( isValid && inLayer){
                  computed = true;
                  int sum=0;
                  int orig = data[tj][tk][tl];
                  int a, b, c;
                  for(int j = tj-1; j <= tj+1; j++){
                      for(int k = tk-1; k <= tk+1; k++){
                          for(int l = tl-1; l<=tl+1; l++){
                              a = j; b = k; c = l;
                              CLAMP_RANGE(a, validJmin, validJmax);
                              CLAMP_RANGE(b, validKmin, validKmax);
                              CLAMP_RANGE(c, validLmin, validLmax);
                              sum += data[a][b][c];
                          }
                      }
                  }
                  sum -= orig;
                  if(orig>0 && (sum <= dieMax || sum >= dieMin))
                      result[tj][tk][tl]=0;
                  else if(orig==0 && (sum >= bornMin && sum <= bornMax))
                      result[tj][tk][tl]=1;
                  else
                      result[tj][tk][tl]=orig;
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                data[tj][tk][tl]= result[tj][tk][tl];
            __syncthreads();
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the
      // small block perform the calculation and switch on ``computed''
      if (computed){
          gpuDataDst[index]=result[tj][tk][tl];
      }
}

/*
   compute N time steps
*/
int simulate(int *gpuData[2],
              int J, int K, int L,
              int timesteps,
              int pyramid_height,
              int blockJ, int blockK, int blockL,
              int border)
{
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
        // TODO: why 3D grid doesn't work??  blockL = 1;
        dim3 dimGrid(blockJ, blockK*blockL);

        int src = 1, dst = 0;
	for (int t = 0; t < timesteps; t+=pyramid_height) {
            int temp = src;
            src = dst;
            dst = temp;
            evolve<<<dimGrid, dimBlock>>>(
                MIN(pyramid_height, timesteps-t),
                gpuData[src], gpuData[dst], J, K, L,
                bornMin, bornMax,
                dieMin,  dieMax,
                blockL, border);
	}
        return dst;
}

int main(int argc, char** argv)
{
    int num_devices;
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) hipSetDevice(DEVICE);

    init(argc, argv);

    run(argc,argv);

    delete [] data;
    delete [] space2D;
    delete [] space3D;

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int border = (pyramid_height)*EXPAND_RATE/2;
    int smallBlockEdge = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int blockJ = J/smallBlockEdge+((J%smallBlockEdge==0)?0:1);
    int blockK = K/smallBlockEdge+((K%smallBlockEdge==0)?0:1);
    int blockL = L/smallBlockEdge+((L%smallBlockEdge==0)?0:1);

    printf("pyramidHeight: %d\ngridSize: [%d, %d, %d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d, %d, %d]\ntargetBlock:[%d]\n",\
	pyramid_height, J, K, L, border, BLOCK_SIZE, blockJ, blockK, blockL, smallBlockEdge);

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

    int *gpuData[2];
    int size = J*K*L;
    hipMalloc((void**)&gpuData[0], sizeof(int)*size);
    hipMalloc((void**)&gpuData[1], sizeof(int)*size);
    hipMemcpy(gpuData[0], data, sizeof(int)*size, hipMemcpyHostToDevice);


    int ret = simulate(gpuData, J, K, L, timesteps,
	 pyramid_height, blockJ, blockK, blockL,
         border);

    hipMemcpy(data, gpuData[ret], sizeof(int)*size, hipMemcpyDeviceToHost);
#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

    printf("Final world:\n");
    for (int i=0; i<J && i<2; i++) {
      for (int j=0; j<K && j<4; j++) {
        for (int k=0; k<L && k<8; k++) {
          printf("%d ", data[k+j*(L)+i*(L*K)]);
        }
        printf("\n");
      }
      printf("\n");
    }

    hipFree(gpuData[0]);
    hipFree(gpuData[1]);
}

