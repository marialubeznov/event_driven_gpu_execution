#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

extern __device__ int* gLocks;
extern __device__ int* gDebugPtr;


template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int numColors, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_color(float* images, float* filters, float* targets,
                                    const int numImages, const int numFilters,
                                    const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                    const int moduleStride, const int numModulesY, const int numModulesX, const int imgStride,
                                    const float scaleTargets, const float scaleOutputs, const int conv);


template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int colorCache, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_sparse(float* images, float* filters, float* targets,
                                       const int numImages, const int numFilters,
                                       const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                       const int moduleStride,
                                       const int numModulesY, const int numModulesX, const int imgStride, const int numImgColors,
                                       const int numGroups,
                                       const float scaleTargets, const float scaleOutputs,
                                       const int conv);

__global__ void emptyKernel()
{

}

/*******************************************************************************/
/*******************************************************************************/
/****************************** GPU Globals ************************************/
/*******************************************************************************/
/*******************************************************************************/

// NOTE: This requires key lengths to be in increments 4 bytes
__device__ int fast_memcmp(const void *key1, const void *key2, int num){

    const unsigned *p1 = (const unsigned* )key1;
    const unsigned *p2 = (const unsigned* )key2;

    int main_loop = num / sizeof(int);

    for(unsigned i=0; i<main_loop; i++){
        if(*(p1+i) != *(p2+i)){
            return 0;
        }
    }

    return 1;
}

/***********************************************/
/***********************************************/
/***********************************************/
#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

////// PREFERS SHARED in config (cudaFuncCachePreferShared)


#define CEIL(x, y) ( (x)/(y) + ( (x)%(y) ? 1 : 0 ) )


template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int numColors, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_color(float* images, float* filters, float* targets,
                                    const int numImages, const int numFilters,
                                    const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                    const int moduleStride, const int numModulesY, const int numModulesX, const int imgStride,
                                    const float scaleTargets, const float scaleOutputs, const int conv) {
    __shared__ float shFilters[B_Y*numColors][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*numColors][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;

    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = blockIdx.y % blocksPerModule;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;
    images += myImgIdx;
    filters += filtersPerThread * B_Y * blockFilterIdx
             + shFilterLoadY * numFilters + shFilterLoadX;
    if (!conv) {
        filters += moduleIdx * numColors * filterPixels * numFilters;
    }

    targets += moduleIdx * numImages
            + (blockFilterIdx * B_Y * filtersPerThread + threadIdx.y) * numImages * numModulesY * numModulesX
            + myImgIdx;


    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }

    for (int p = 0; p < filterPixels; p += B_Y) {
        /*
         * Load B_Y pixels from B_Y*filtersPerThread filters
         */
        if (shFilterLoadY < B_Y) {
            #pragma unroll
            for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                if (p + p2 + shFilterLoadY < filterPixels) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[(c * filterPixels + p + p2) * numFilters];
                    }
                } else {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                    }
                }
            }
        }

        /*
         * Load B_Y pixels from B_X*imgsPerThread images
         */
        const int pixIdx = p + threadIdx.y;
        if (pixIdx < filterPixels) {
            const int x = paddingStart + imgLoadModPosX + pixIdx % filterSize;
            const int y = paddingStart + imgLoadModPosY + pixIdx / filterSize;
            if (y >= 0 && y< imgSizeY && x >= 0 && x < imgSizeX) {
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = images[imgStride * (c * imgPixels + y * imgSizeX + x) + i * B_X];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            } else { // Padding
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                    }
                }
            }
        }
        __syncthreads();
        #pragma unroll
        for (int i = 0; i < B_Y*numColors; i++) {
            #pragma unroll
            for(int f = 0; f < filtersPerThread; f++) {
                #pragma unroll
                for(int g = 0; g < imgsPerThread; g++) {
                    prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                }
            }

        }
        __syncthreads();
    }

    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] = scaleOutputs * prod[f][g];
                }
            }
        }
    }


}


template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int colorCache, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_sparse(float* images, float* filters, float* targets,
                                       const int numImages, const int numFilters,
                                       const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                       const int moduleStride,
                                       const int numModulesY, const int numModulesX, const int imgStride, const int numImgColors,
                                       const int numGroups,
                                       const float scaleTargets, const float scaleOutputs,
                                       const int conv) {
    __shared__ float shFilters[B_Y*colorCache][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*colorCache][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;
    const int numFilterColors = numImgColors / numGroups;
    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = filtersPerThread * B_Y * (blockIdx.y % blocksPerModule);
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockGroupIdx = blockFilterIdx / numFiltersPerGroup;

    const int numModules = numModulesX * numModulesY;
    const int blockColorIdx = numFilterColors * blockGroupIdx;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = paddingStart + (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = paddingStart + (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;

    images += blockColorIdx * imgPixels * imgStride + myImgIdx;
    filters +=blockFilterIdx
            + shFilterLoadY * numFilters + shFilterLoadX;
    if (!conv) {
        filters += moduleIdx * numFilterColors * filterPixels * numFilters;
    }

    targets += moduleIdx * numImages
            + (blockFilterIdx + threadIdx.y) * numImages * numModules
            + myImgIdx;

    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }
//    __shared__ int imgPos[]
    for (int oc = 0; oc < numFilterColors; oc += colorCache) { // oc stands for outer color (loop)
        for (int p = 0; p < filterPixels; p += B_Y) {
            /*
             * Load B_Y pixels from B_Y*filtersPerThread filters
             */
            if (shFilterLoadY < B_Y) {
                #pragma unroll
                for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                    if (p + p2 + shFilterLoadY < filterPixels) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[((oc+c) * filterPixels + p + p2) * numFilters];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                        }
                    }
                }
            }

            /*
             * Load B_Y pixels from B_X*imgsPerThread images
             */
            const int pixIdx = p + threadIdx.y;
            if (pixIdx < filterPixels) {
                const int x = imgLoadModPosX + pixIdx % filterSize;
                const int y = imgLoadModPosY + pixIdx / filterSize;
                if (y >= 0 && y < imgSizeY && x >= 0 && x < imgSizeX) {
                    float* m = &images[imgStride * (oc * imgPixels + y * imgSizeX + x)];
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = m[c * imgStride * imgPixels + i * B_X];
                            }
                        } else {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                            }
                        }
                    }
                } else { // Padding
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            }
            __syncthreads();
            #pragma unroll
            for (int i = 0; i < B_Y*colorCache; i++) {
                #pragma unroll
                for(int f = 0; f < filtersPerThread; f++) {
                    #pragma unroll
                    for(int g = 0; g < imgsPerThread; g++) {
                        prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                    }
                }

            }
            __syncthreads();
        }
    }

    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModules] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleOutputs * prod[f][g];
                }
            }
        }
    }


}

void _filterActs(float *images, int images_cols, int images_rows, float *filters, int filters_cols, 
                int filters_rows,  float *targets, int targets_cols, int targets_rows,
                int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                int numImgColors, int numGroups, float scaleTargets, float scaleOutput, int conv, hipStream_t stream, 
                bool warmup) {

    int numFilterColors = numImgColors / numGroups;      
    int numFilters = filters_cols;
    int numModules = numModulesY * numModulesX;
    int numImages = images_cols;
    int imgPixels = images_rows/numImgColors;
    int imgSizeX = imgPixels / imgSizeY;
    int filterModuleMult = conv ? 1 : numModules;
    
    assert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)));
    assert(numGroups == 1 || numFilterColors % 2 == 0);
    assert(numFilters % (16 * numGroups) == 0);
    assert(numImgColors % numGroups == 0);
    assert(images_rows == imgPixels * numImgColors);
    assert(imgSizeY * imgSizeX == imgPixels);
    int numFiltersPerGroup = numFilters / numGroups;

    int imgStride = images_cols; // ???? //images.getStride(); // images does not need to be a contiguous matrix

    int filterPixels = filters_rows / (filterModuleMult * numFilterColors);
    int filterSize = int(sqrt(filterPixels));
    assert(filterSize * filterSize == filterPixels);
    assert(filters_rows == filterModuleMult * numFilterColors * filterPixels);

    // These routines don't handle the case when only part of the image is visited in the convolution
    assert(paddingStart <= 0);
    assert(paddingStart + (numModulesX-1)*moduleStride + filterSize >= imgSizeX);
    assert(paddingStart + (numModulesY-1)*moduleStride + filterSize >= imgSizeY);
    assert(moduleStride <= filterSize);
    
    int imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;
    dim3 blocks = numFiltersPerGroup % 32 == 0 ? dim3(DIVUP(numImages, 32 * imgsPerThread), (numModules * numFilters) / (4 * 8))
                                               : dim3(DIVUP(numImages, 32 * imgsPerThread), (numModules * numFilters) / (4 * 4));

    if( warmup ) {
        blocks = dim3(4, 16);
    }

    dim3 threads(32, 4);
    bool checkImgBounds = numImages % (32*imgsPerThread) != 0;

    printf("blocks(%d, %d, %d), threads(%d, %d, %d)\n", blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    
    /*
    if (scaleTargets == 0) {
        targets.resize(numFilters * numModules, numImages);
    } else {
        assert(targets.getNumRows() == numFilters * numModules);
        assert(targets.getNumCols() == numImages);
    }
    */


    assert(targets_rows == numFilters * numModules);
    assert(targets_cols == numImages);

    printf("\n\n");
    printf("filters.getNumCols = %d, filters.getnumrows = %d, images.getNumCols = %d, images.getNumRows = %d, targets.getNumcols = %d, targets.getNumrows = %d\n\n",
            filters_cols, filters_rows, images_cols, images_rows, targets_cols, targets_rows);

    printf("\n\n\n====== Kernel Parameters ======\n\n");

    printf("images = %p\n"
        "filters = %p\n"
        "targets = %p\n"
        "numImages = %d\n"
        "numFilters = %d\n"
        "imgSizeY = %d\n"
        "imgSizeX = %d\n"
        "filterSize = %d\n"
        "paddingStart = %d\n"
        "moduleStride = %d\n"
        "numModulesY = %d\n"
        "numModulesX = %d\n"
        "imgStride = %d\n"
        "scaleTargts = %lf\n"
        "scaleOutputs = %lf\n"
        "conv = %d\n"
        "numImgColors = %d\n"
        "imgsPerThread = %d\n"
        "numGroups = %d\n"
        "checkImgBounds = %d\n"
        "numFiltersPerGroup = %d\n"
        "blocks = %d, %d, %d\n"
        "threads = %d, %d, %d\n"
        "\n===================================\n",
        images, filters, targets,
        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart,
        moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv,
        numImgColors, imgsPerThread, numGroups, checkImgBounds, numFiltersPerGroup, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);

#if 0    
    dim3 tmpBlocks(4, 64, 1);
        //filterActs_YxX_color < 4, 32, 1, 4, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
        filterActs_YxX_color<<<tmpBlocks, threads, 0, stream>>>(images, filters, targets, numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, 
#endif

    if (imgsPerThread == 4) {
        if (numImgColors <= 3) {
            assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            ////cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 1, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 1, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 1, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 1, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 2, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 2, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 2, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 2, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 3, false, true >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 8, 3, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 3, false, true >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 4, 3, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 3, false, false >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 8, 3, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 3, false, false >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 4, 3, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 1, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 1, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 1, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 1, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 2, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 2, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 2, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 2, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 3, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 3, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 3, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 3, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 8, 3, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 3, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 4, 4, 3, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 3, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 8, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 4, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 8, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 4, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 8, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 4, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 8, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 4, 4, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    } else if (imgsPerThread == 2) {
        if (numImgColors <= 3) {
            assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 1, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 1, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 1, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 1, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 2, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 2, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 2, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 2, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 3, false, true >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 8, 3, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 3, false, true >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 4, 3, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 3, false, false >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 8, 3, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 3, false, false >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 4, 3, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 1, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 1, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 1, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 1, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 2, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 2, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 2, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 2, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 3, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 3, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 3, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 3, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 8, 3, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 3, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 2, 4, 3, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 3, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 8, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 4, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 8, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 4, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 8, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 4, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 8, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 2, 4, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    } else {
        if (numImgColors <= 3) {
            assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 1, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            printf("\n\n\n\ I AM HERE \n\n\n");
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 1, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 1, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 1, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 2, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 2, false, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 2, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 2, false, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 3, false, true >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 8, 3, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             printf("\n\n\n\nBING HERE\n\n\n\n");
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 3, false, true >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 4, 3, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 3, false, false >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 8, 3, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 3, false, false >, cudaFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 4, 3, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 1, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 1, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 1, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 1, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 2, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 2, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 2, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 2, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 3, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 3, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 3, true, true >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 3, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 8, 3, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 3, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            //cudaFuncSetCacheConfig(filterActs_YxX_color< 4, 32, 1, 4, 3, true, false >, cudaFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 3, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 8, 2, false, true >, cudaFuncCachePreferShared);
                        printf("\n\n\n\n\n BING BING BING \n\n\n\n\n");
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 4, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, false, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 8, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 4, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, false, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 8, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 4, 2, false, true >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, true, true > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 8, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        //cudaFuncSetCacheConfig(filterActs_YxX_sparse< 4, 32, 1, 4, 2, false, false >, cudaFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, true, false > <<<blocks, threads, 0, stream>>>(images, filters, targets,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    }

}

  

